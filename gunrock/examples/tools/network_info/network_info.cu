#include "hip/hip_runtime.h"
#include <gunrock/algorithms/algorithms.hxx>
#include <gunrock/algorithms/tc.hxx>
// Reference implementations
using namespace gunrock;
using namespace memory;

void test_network_info(int num_arguments, char** argument_array) {
	if (num_arguments < 2) {
		std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
		exit(1);
	}

	// Define types
	using vertex_t = int;
	using edge_t = int;
	using weight_t = float;
	using csr_t = format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

	// IO
	csr_t csr;
	std::string filename = argument_array[1];
	// std::cout << filename  << std::endl;
	if (util::is_market(filename)) {
		io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
		csr.from_coo(mm.load(filename));
	} else if (util::is_binary_csr(filename)) {
		csr.read_binary(filename);
	} else {
		std::cerr << "Unknown file format: " << filename << std::endl;
		exit(1);
	}

	// Build graph
	// supports row_indices and column_offsets (default = nullptr)
	auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
		csr.number_of_rows,               // rows
		csr.number_of_columns,            // columns
		csr.number_of_nonzeros,           // nonzeros
		csr.row_offsets.data().get(),     // row_offsets
		csr.column_indices.data().get(),  // column_indices
		csr.nonzero_values.data().get()   // values
	);
	
	// Vertices and Edges
	long int n_vertices = G.get_number_of_vertices();
	long int n_edges = G.get_number_of_edges();
	// Density Evaluation
	double density = (double)n_edges / (double)(n_vertices*(n_vertices-1));
	
	// Degree Metrics Evalutations
	// Copy to CPU
	thrust::host_vector<edge_t> _row_offsets(csr.row_offsets);
	edge_t* row_offsets = _row_offsets.data();
	int min_degree = std::numeric_limits<int>::max();
	int max_degree = std::numeric_limits<int>::min();
	for(vertex_t i = 0; i < n_vertices; i++){
		double degree = row_offsets[i+1] - row_offsets[i];
		if(degree < min_degree) min_degree = degree;
		if(degree > max_degree) max_degree = degree;
	}
	double avg_degree = (double)n_edges / (double)n_vertices;
	double avg_degree_variance = 0;
	for(vertex_t i = 0; i < n_vertices; i++){
		double degree = row_offsets[i+1] - row_offsets[i];
		avg_degree_variance += ((degree - avg_degree) * (degree - avg_degree));
	}
	avg_degree_variance /= n_vertices;
	// Clustering Coefficients Evaluations
	// Triangle Count
	thrust::device_vector<vertex_t> triangles_count(G.get_number_of_vertices(), 0);
	std::size_t total_triangles = 0;
	tc::run(G, true,triangles_count.data().get(), &total_triangles);
	// Copy to CPU
	thrust::host_vector<vertex_t> _triangles_count(triangles_count);
	vertex_t* triangles = _triangles_count.data();
	double C_global = 0;
	double C_avg = 0;
	for(vertex_t i = 0; i < n_vertices; i++){
		double degree = row_offsets[i+1] - row_offsets[i];
		C_global += degree * (degree - 1);
		if(degree != 0 && degree != 1)
			C_avg += (triangles[i] * 2) / (degree * (degree - 1));
	}
	C_global = (total_triangles / C_global) * 2;
	C_avg = C_avg / n_vertices;
	
	// Print
	std::cout << "Vertices,Edges,Density,Maximum degree,Minimum degree,Average degree,Average degreee variance,Global clustering coefficient,Average clustering coefficient" << std::endl;
	// Vertices and Edges
	std::cout << n_vertices << "," << n_edges << ",";
	// Density
	std::cout << density << ",";
	// Maximum degree and Minimum Degree
	std::cout << max_degree << "," << min_degree << ",";
	// Average Degree
	std::cout << avg_degree << ",";
	// Average Degree Variance
	std::cout << avg_degree_variance << ",";
	// Global Clustering Coefficients
	std::cout << C_global<< ",";
	// Average Clustering Coefficients
	std::cout << C_avg << std::endl;
}

int main(int argc, char** argv) {
	test_network_info(argc, argv);
}