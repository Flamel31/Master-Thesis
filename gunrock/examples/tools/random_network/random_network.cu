#include <gunrock/algorithms/algorithms.hxx>

using namespace gunrock;
using namespace memory;

void test_random_network(int num_arguments, char** argument_array) {
	if (num_arguments != 4) {
		std::cerr << "usage: ./bin/<program-name> num_vertices probability_ij folder_path" << std::endl;
		exit(1);
	}
	
	int num_vertices = atoi(argument_array[1]);
	float probability_ij = atof(argument_array[2]);
	
	if (probability_ij < 0 || probability_ij > 1) {
		std::cerr << "probability_ij should be between 0.0 and 1.0" << std::endl;
		exit(1);
	}
	
	// Define types
	using vertex_t = int;
	using edge_t = int;
	using weight_t = float;

	// IO
	std::string outpath = argument_array[3];
	// Setting random seed
	srand(time(NULL));

	using csr_t = format::csr_t<memory::memory_space_t::host, vertex_t, edge_t, weight_t>;
	csr_t csr;
	
	int last_count = 0;
	int count = 0;
	csr.row_offsets.push_back(count);
	for(vertex_t i = 0; i < num_vertices ; i++){
		last_count += count;
		count = 0;
		for(vertex_t j = 0; j < num_vertices ; j++){
			if(i == j) continue;
			// Random Value between 0.0 and 1.0
			float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
			if(r <= probability_ij){
				// Random Weight between 1.0 and 64.0
				weight_t w = 1 + static_cast <weight_t> (rand()) /( static_cast <weight_t> (RAND_MAX/(63)));
				csr.column_indices.push_back(j);
				csr.nonzero_values.push_back(w);
				count++;
			}			
		}
		csr.row_offsets.push_back(last_count+count);
	}
	
	csr.number_of_rows = num_vertices;
	csr.number_of_columns = num_vertices;
	csr.number_of_nonzeros = csr.nonzero_values.size();

	std::cout << "Generating network with " << num_vertices << " vertices and probability of an edge between i and j of " << probability_ij << std::endl;
	std::cout << "csr.number_of_rows     = " << csr.number_of_rows << std::endl;
	std::cout << "csr.number_of_columns  = " << csr.number_of_columns << std::endl;
	std::cout << "csr.number_of_nonzeros = " << csr.number_of_nonzeros << std::endl;
	std::cout << "writing to             = " << outpath << std::endl;

	csr.write_binary(outpath);
}

int main(int argc, char** argv) {
	test_random_network(argc, argv);
}